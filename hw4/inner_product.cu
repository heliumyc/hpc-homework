//
// Created by CONG YU on 4/20/20.
//

#include <hip/hip_runtime.h>
#include <algorithm>
#include <stdio.h>
#include <omp.h>
#include <random>
#include <string>

void Check_CUDA_Error(const char *message) {
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        fprintf(stderr, "ERROR: %s: %s\n", message, hipGetErrorString(error));
        exit(-1);
    }
}

// cpu sequential computation
void sequential_vec_inner_product(double *res, const double *a, const double *b, long n) {
    double acc = 0;
    for (long i = 0; i < n; i++) {
        acc += a[i]*b[i];
    }
    *res = acc;
}

// openmp cpu map
void openmp_map_vec_inner_product(const double *a, const double *b, double *c, long n) {
#pragma omp parallel for schedule(static)
    for (long i = 0; i < n; i++) {
        c[i] = a[i]*b[i];
    }
}

// openmp cpu reduce
void openmp_reduce_sum(double *res, const double *c, long n) {
    double sum = 0;
//    omp_set_num_threads(6);
#pragma omp parallel for reduction (+: sum)
    for (long i = 0; i < n; i++) {
        sum += c[i];
    }
    *res = sum;
}

#define BLOCK_SIZE 1024

// gpu reduce
__global__
void gpu_map_vec_inner_product(const double*a, const double *b, double *c, long n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        c[idx] = a[idx] * b[idx];
    }
}

// gpu reduce
__global__
void gpu_reduce_inner_product_kernal1(const double *a, double *sum, long n) {
    __shared__ double smem[BLOCK_SIZE];
    int idx = (blockIdx.x) * blockDim.x + threadIdx.x;

    // each thread reads data from global into shared memory
    if (idx < n) smem[threadIdx.x] = a[idx];
    else smem[threadIdx.x] = 0;
    __syncthreads();

    // x >>= 1 means "set x to itself shifted by one bit to the right", i.e., a divison by 2
    // write to memory with threadIdx rather than ``index''
    for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
        if (threadIdx.x < s) {
            smem[threadIdx.x] += smem[threadIdx.x + s];
        }
        __syncthreads();
    }

    // write to global memory
    if (threadIdx.x == 0) sum[blockIdx.x] = smem[threadIdx.x];

}

__global__ void reduction_kernel0(double* sum, const double* a, long N){
    __shared__ double smem[BLOCK_SIZE];
    int idx = (blockIdx.x) * blockDim.x + threadIdx.x;

    // each thread reads data from global into shared memory
    if (idx < N) smem[threadIdx.x] = a[idx];
    else smem[threadIdx.x] = 0;
    __syncthreads();

    for(int s = 1; s < blockDim.x; s *= 2) {
        if(threadIdx.x % (2*s) == 0)
            smem[threadIdx.x] += smem[threadIdx.x + s];
        __syncthreads();
    }

    // write to global memory
    if (threadIdx.x == 0) sum[blockIdx.x] = smem[threadIdx.x];
}

int main() {
    long n = (1UL<<25); // 2^25

    // malloc
//    auto* a = (double*) malloc(n * sizeof(double));
//    auto* b = (double*) malloc(n * sizeof(double));
//    auto* temp = (double*) malloc(n * sizeof(double));
    double* a;
    double* b;
    double* temp;
    hipHostMalloc((void**)&a, n * sizeof(double), hipHostMallocDefault);
    hipHostMalloc((void**)&b, n * sizeof(double), hipHostMallocDefault);
    hipHostMalloc((void**)&temp, n * sizeof(double), hipHostMallocDefault);

    // random
    std::random_device rd;
    std::default_random_engine gen(rd());
    std::uniform_real_distribution<double> uniformRealDistribution(-1, 1);

    // init
    omp_set_num_threads(6);
#pragma omp parallel for schedule(static)
    for (long i = 0; i< n; i++) {
        a[i] = uniformRealDistribution(gen);
        b[i] = uniformRealDistribution(gen);
    }

    double time;

    // sequential calculation
    double tick = omp_get_wtime();
    double ref;
    sequential_vec_inner_product(&ref, a, b, n);
    time = omp_get_wtime() - tick;
    printf("Sequential benchmark\n");
    printf("Time = %f\n", time/1e9);
    printf("CPU Bandwidth = %f GB/s\n", 2*n*sizeof(double) / time/1e9);
    printf("Error = %f\n", std::abs(ref-ref));

    printf("------------\n");

    // openmp calculation
    tick = omp_get_wtime();
    openmp_map_vec_inner_product(a, b, temp, n);
    double openmp_res;
    openmp_reduce_sum(&openmp_res, temp, n);
    time = omp_get_wtime() - tick;
    printf("Openmp benchmark\n");
    printf("Time = %f\n", time/1e9);
    printf("CPU Bandwidth = %f GB/s\n", 4*n*sizeof(double) / time/1e9);
    printf("Error = %f\n", std::abs(openmp_res-ref));

    printf("------------\n");

    // cuda

    // init
    double* a_d;
    double* b_d;
    hipMalloc(&a_d, n*sizeof(double));
    hipMalloc(&b_d, n*sizeof(double));
    hipMemcpyAsync(a_d, a, n*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(b_d, b, n*sizeof(double), hipMemcpyHostToDevice);

    double* temp_d;
    hipMalloc(&temp_d, n*sizeof(double));
    double* extra_d;
    long N_work = 1;
    for (long i = (n+BLOCK_SIZE-1)/(BLOCK_SIZE); i > 1; i = (i+BLOCK_SIZE-1)/(BLOCK_SIZE)) N_work += i;
    hipMalloc(&extra_d, N_work*sizeof(double)); // extra memory buffer for reduction across thread-blocks
    hipDeviceSynchronize();

    tick = omp_get_wtime();
    double cuda_res;

    gpu_map_vec_inner_product<<<n/BLOCK_SIZE,BLOCK_SIZE>>>(a_d, b_d, temp_d, n);

    double* sum_d = extra_d;
    long Nb = (n+BLOCK_SIZE-1)/(BLOCK_SIZE);
    reduction_kernel0<<<Nb,BLOCK_SIZE>>>(sum_d, temp_d, n);
    while (Nb > 1) {
        long lastN = Nb;
        Nb = (Nb+BLOCK_SIZE-1)/(BLOCK_SIZE);
        reduction_kernel0<<<Nb,BLOCK_SIZE>>>(sum_d + lastN, sum_d, lastN);
        sum_d += lastN;
    }
    hipMemcpyAsync(&cuda_res, sum_d, 1*sizeof(double), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    time = omp_get_wtime() - tick;
    printf("GPU benchmark\n");
    printf("Time = %f\n", time/1e9);
    printf("CPU Bandwidth = %f GB/s\n", 4*n*sizeof(double) / time/1e9);
    printf("Error = %f\n", std::abs(cuda_res-ref));

    // free
    hipHostFree(a);
    hipHostFree(b);
    hipHostFree(temp);
    hipFree(a_d);
    hipFree(b_d);
    hipFree(temp_d);
    hipFree(extra_d);
}
