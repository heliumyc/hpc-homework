
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <omp.h>
#include <algorithm>

int N = 100;
int SIZE = N+2; // always N+2
int MAT_SIZE = SIZE*SIZE;
//long maxIter = INT32_MAX;
long maxIter = 30000;
double h, hSqr, hSqrInverse;

inline double sqr(double x) {
    return x*x;
}

void Check_CUDA_Error(const char *message) {
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        fprintf(stderr, "ERROR: %s: %s\n", message, hipGetErrorString(error));
        exit(-1);
    }
}

double calcResidual(const double* u) {
    // since f is always 1, just hardcoded 1 into formula
    // N is actually N+2 (0-N+1)
    double res = 0;
    for (int i = 1; i <= N; ++i) {
        for (int j = 1; j <= N; ++j) {
            res += sqr((-u[(i-1)*SIZE+j]-u[i*SIZE+j-1]+4*u[i*SIZE+j]-u[(i+1)*SIZE+j]-u[i*SIZE+j+1]) * hSqrInverse - 1);
        }
    }
    return std::sqrt(res);
}

/**
 * return the iteration of jacobi
 * @return
 */
long jacobi_cpu(double* u, double* v) {
    double initResidual = calcResidual(u);
    double curResidual = 0;
    long k;
    for (k = 1; k <= maxIter; ++k) {
#   pragma omp parallel for num_threads(6)
        for (int i = 1; i <= N; ++i) {
            for (int j = 1; j <= N; ++j) {
                // update u
                v[i*SIZE+j] = (hSqr+u[(i-1)*SIZE+j]+u[i*SIZE+j-1]+u[(i+1)*SIZE+j]+u[i*SIZE+j+1])/4;
            }
        }
        std::swap(u, v);
        curResidual = calcResidual(u);

        if (initResidual/curResidual > 1e+6) {
            break;
        }
    }
    return k;
}

#define TILE_LEN 16 // block size be 8*8=64

__device__ double gpu_residual;

__device__ double atomicAdd2(double* address, double val)
{
    auto* address_as_ull =
            (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                                             __longlong_as_double(assumed)));

        // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}

__global__ void gpu_residual_calc(const double* u, int n, double _hsqrinverse) {
    __shared__ double smem[TILE_LEN][TILE_LEN];
    int i = (threadIdx.x + 1) + blockIdx.x*blockDim.x;
    int j = (threadIdx.y + 1) + blockIdx.y*blockDim.y;

    int size = n+2;
    if(i <= n && j <= n){
        double diff = (-u[(i-1)*size+j]-u[i*size+j-1]+4*u[i*size+j]-u[(i+1)*size+j]-u[i*size+j+1]) * _hsqrinverse - 1;
        diff = diff*diff;
        atomicAdd2(&gpu_residual, 1);
//        smem[threadIdx.x][threadIdx.y] = diff;
//        __syncthreads();
    }

//    if (threadIdx.x == 0 && threadIdx.y == 0) {
//        double acc = 0;
//        for (int k = 0; k < TILE_LEN; k++) {
//            for (int p = 0; p < TILE_LEN; p++) {
//                acc += smem[p][k];
//            }
//        }
//        atomicAdd2(&gpu_residual, acc);
//    }

//    if (threadIdx.y == 0) {
//        double acc = 0;
//        for (int k = 0; k < TILE_LEN; k++) {
//            acc += smem[threadIdx.x][k];
//        }
//        smem[threadIdx.x][0] = acc;
//        __syncthreads();
//    }
//
//    if (threadIdx.x == 0 && threadIdx.y == 0) {
//        double acc = 0;
//        for (int k = 0; k < TILE_LEN; k++) {
//            acc += smem[k][0];
//        }
//        atomicAdd2(&gpu_residual, acc);
//    }
}

__global__ void gpu_jacobi(double* u, double* v, double hsqr, int size) {
    int i = (threadIdx.x + 1) + blockIdx.x*blockDim.x;
    int j = (threadIdx.y + 1) + blockIdx.y*blockDim.y;
    v[i*size+j] = (hsqr+u[(i-1)*size+j]+u[i*size+j-1]+u[(i+1)*size+j]+u[i*size+j+1])/4;
}


int main(int argc, char** argv) {
    printf("Jacobi 2D\n");
    printf("=====================\n");

    h = 1./(double) (N+1);
    hSqr = h*h;
    hSqrInverse = 1/hSqr;

    auto* u = new double[MAT_SIZE];
    auto* v = new double[MAT_SIZE];
    // initialization
    for (int i = 0; i < SIZE*SIZE; ++i) {
        u[i] = 0;
        v[i] = 0;
    }

    double tick = omp_get_wtime();
    double tok;
    // jacobi start
    long cpu_iter = jacobi_cpu(u, v);
    tok = omp_get_wtime();
    printf("Openmp cpu\n");
    printf("Used time: %lf \n Iteration: %ld\n", (tok-tick), cpu_iter);

    printf("=====================\n");

    // gpu

//    // allocate
    for (int i = 0; i < SIZE*SIZE; ++i) {
        u[i] = 0;
        v[i] = 0;
    }
    printf("test stop");
    printf("%lf", u[0]);
    double* u_d;
    double* v_d;
    hipMalloc(&u_d, MAT_SIZE * sizeof(double));
    hipMalloc(&v_d, MAT_SIZE * sizeof(double));
    hipMemcpyAsync(u_d, u, MAT_SIZE * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(v_d, v, MAT_SIZE * sizeof(double), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    Check_CUDA_Error("alloc failed");

    dim3 grid((N+TILE_LEN-1)/TILE_LEN, (N+TILE_LEN-1)/TILE_LEN);
    dim3 block(TILE_LEN, TILE_LEN);

    printf("test stop");
    tick = omp_get_wtime();
    long gpu_iter = 0;
    double init_res = 0;

    hipMemcpyToSymbol(HIP_SYMBOL(gpu_residual), &init_res, sizeof(double)); // load to gpu global var
    hipMemcpyFromSymbol(&init_res, HIP_SYMBOL(gpu_residual), sizeof(double)); // load back to init residual
    printf("test %f", init_res);
    Check_CUDA_Error("init failed");
    hipDeviceSynchronize();
    gpu_residual_calc<<<grid, block>>>(u_d, N, hSqrInverse);
    hipMemcpyFromSymbol(&init_res, HIP_SYMBOL(gpu_residual), sizeof(double)); // load back to init residual

    hipDeviceSynchronize();
    printf("%f", init_res);
    double cur_res = 0;
    while (gpu_iter < maxIter) {
        gpu_jacobi<<<grid, block>>>(u_d, v_d, N, hSqr);
        std::swap(u_d, v_d);
        gpu_residual_calc<<<grid, block>>>(u_d, N, hSqrInverse);
        hipMemcpyFromSymbol(&cur_res, HIP_SYMBOL(gpu_residual), sizeof(double));
        hipDeviceSynchronize();
        if (init_res/cur_res > 1e+6) {
            break;
        }
        gpu_iter++;
    }

    tok = omp_get_wtime();
    printf("GPU\n");
    printf("Used time: %lf \n Iteration: %ld\n", (tok-tick), gpu_iter);
    printf("Residual: %lf\n", cur_res);

    free(u);
    free(v);
    hipFree(u_d);
    hipFree(v_d);
}
