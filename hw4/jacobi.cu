
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <omp.h>
#include <algorithm>

int N = 200;
int SIZE = N+2; // always N+2
int MAT_SIZE = SIZE*SIZE;
long maxIter = INT32_MAX;
double h, hSqr, hSqrInverse;

inline double sqr(double x) {
    return x*x;
}

double calcResidual(const double* u) {
    // since f is always 1, just hardcoded 1 into formula
    // N is actually N+2 (0-N+1)
    double res = 0;
    for (int i = 1; i <= N; ++i) {
        for (int j = 1; j <= N; ++j) {
            res += sqr((-u[(i-1)*SIZE+j]-u[i*SIZE+j-1]+4*u[i*SIZE+j]-u[(i+1)*SIZE+j]-u[i*SIZE+j+1]) * hSqrInverse - 1);
        }
    }
    return std::sqrt(res);
}

/**
 * return the iteration of jacobi
 * @return
 */
long jacobi_cpu(double* u, double* v) {
    double initResidual = calcResidual(u);
    double curResidual = 0;
    long k;
    for (k = 1; k <= maxIter; ++k) {
#   pragma omp parallel for num_threads(6)
        for (int i = 1; i <= N; ++i) {
            for (int j = 1; j <= N; ++j) {
                // update u
                v[i*SIZE+j] = (hSqr+u[(i-1)*SIZE+j]+u[i*SIZE+j-1]+u[(i+1)*SIZE+j]+u[i*SIZE+j+1])/4;
            }
        }
        std::swap(u, v);
        curResidual = calcResidual(u);

        if (initResidual/curResidual > 1e+6) {
            break;
        }
    }
    return k;
}

#define TILE_LEN 16 // block size be 8*8=64

__device__ double gpu_residual;

__device__ double atomicAdd2(double* address, double val)
{
    auto* address_as_ull =
            (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                                             __longlong_as_double(assumed)));

        // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}

__global__ void gpu_residual_calc(const double* u, int n, double _hsqrinverse) {
    __shared__ double smem[TILE_LEN][TILE_LEN];
    int i = (threadIdx.x + 1) + blockIdx.x*blockDim.x;
    int j = (threadIdx.y + 1) + blockIdx.y*blockDim.y;

    int size = n-2;
    if(i <= n && j <= n){
        double diff = (-u[(i-1)*size+j]-u[i*size+j-1]+4*u[i*size+j]-u[(i+1)*size+j]-u[i*size+j+1]) * _hsqrinverse - 1;
        diff = std::sqrt(diff);
        smem[threadIdx.x][threadIdx.y] = diff;
        __syncthreads();
    }

    if (threadIdx.y == 0) {
        double acc = 0;
        for (int k = 0; k < TILE_LEN; k++) {
            acc += smem[threadIdx.x][threadIdx.y];
        }
        smem[threadIdx.x][0] = acc;
        __syncthreads();
    }

    if (threadIdx.x == 0 && threadIdx.y == 0) {
        double acc = 0;
        for (int k = 0; k < TILE_LEN; k++) {
            acc += smem[threadIdx.x][0];
        }
        atomicAdd2(&gpu_residual, acc);
    }
}

__global__ void gpu_jacobi(double* u, double* v, double hsqr, int size) {
    int i = (threadIdx.x + 1) + blockIdx.x*blockDim.x;
    int j = (threadIdx.y + 1) + blockIdx.y*blockDim.y;
    v[i*size+j] = (hsqr+u[(i-1)*size+j]+u[i*size+j-1]+u[(i+1)*size+j]+u[i*size+j+1])/4;
}


int main(int argc, char** argv) {
    printf("Jacobi 2D\n");
    printf("=====================\n");

    h = 1./(double) (N+1);
    hSqr = h*h;
    hSqrInverse = 1/hSqr;

    auto* u = new double[MAT_SIZE];
    auto* v = new double[MAT_SIZE];
    // initialization
    for (int i = 0; i < SIZE*SIZE; ++i) {
        u[i] = 0;
        v[i] = 0;
    }

    double tick = omp_get_wtime();
    double tok;
    // jacobi start
    long cpu_iter = jacobi_cpu(u, v);
    tok = omp_get_wtime();
    printf("Openmp cpu\n");
    printf("Used time: %lf \n Iteration: %ld\n", (tok-tick), cpu_iter);

    printf("=====================\n");

    // gpu

//    // allocate
    std::fill(u, u+MAT_SIZE, 0);
    std::fill(v, v+MAT_SIZE, 0);
    double* u_d;
    double* v_d;
    hipMalloc(&u_d, MAT_SIZE * sizeof(double));
    hipMalloc(&v_d, MAT_SIZE * sizeof(double));
    hipMemcpyAsync(u_d, u, MAT_SIZE * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(v_d, v, MAT_SIZE * sizeof(double), hipMemcpyHostToDevice);

    dim3 grid(N/TILE_LEN, N/TILE_LEN);
    dim3 block(TILE_LEN, TILE_LEN);

    tick = omp_get_wtime();
    long gpu_iter = 0;
    double init_res;
    gpu_residual_calc<<<grid, block>>>(u_d, N, hSqrInverse);
    hipMemcpyToSymbol(HIP_SYMBOL(gpu_residual), &init_res, sizeof(double));
    double cur_res = 0;
    while (gpu_iter < maxIter) {
        gpu_jacobi<<<grid, block>>>(u_d, v_d, N, hSqr);
        std::swap(u_d, v_d);
        gpu_residual_calc<<<grid, block>>>(u_d, N, hSqrInverse);
        hipMemcpyToSymbol(HIP_SYMBOL(gpu_residual), &cur_res, sizeof(double));
        if (init_res/cur_res > 1e+6) {
            break;
        }
    }

    tok = omp_get_wtime();
    printf("GPU\n");
    printf("Used time: %lf \n Iteration: %ld\n", (tok-tick), gpu_iter);

    free(u);
    free(v);
    hipFree(u_d);
    hipFree(v_d);
}